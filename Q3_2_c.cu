#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <iomanip>
// Kernel function to add the elements of two arrays
__global__
void add(int n, float *x, float *y)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride)
    y[i] = x[i] + y[i];
}


int main(int argc, char** argv)
{
  int K = atoi(argv[1]) * 1000000;
  size_t size = K * sizeof(float);

  //Initialize host side arrays and allocate memory

  float *h_x = (float*)malloc(K * sizeof(float));
  float *h_y = (float*)malloc(K * sizeof(float));

  for (int i = 0; i < K; i++) {
    h_x[i] = 1.0f;
    h_y[i] = 2.0f;
  }
  
  //Initialize device side arrays and allocate memory in CUDA
  float *d_x, *d_y;
  hipMalloc((void**)&d_x, size);
  hipMalloc((void**)&d_y, size);
  
  //Copy arrays to device
  hipMemcpy(d_x, h_x, size, hipMemcpyHostToDevice);
  hipMemcpy(d_y, h_y, size, hipMemcpyHostToDevice);

  // Run kernel on KM elements on the GPU
  int blockSize = 256;
  int numBlocks = (K + blockSize - 1) / blockSize;
  add<<<numBlocks, blockSize>>>(K, x, y);

 //Copy results to host side array
  hipMemcpy(h_y, d_y, size, hipMemcpyDeviceToHost);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (int i = 0; i < K; i++)
    maxError = fmax(maxError, fabs(h_y[i]-3.0f));
  std::cout << "Max error: " << maxError << std::endl;

  // Free memory
  hipFree(d_x);
  hipFree(d_y);
  free(h_x);
  free(h_y);
  
  return 0;
}